// mandel_skeleton.cu by Ruben Spaans and Anne C. Elster BSD v2 license
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
/* Problem size */
#define XSIZE 2560
#define YSIZE 2048

/* Divide the problem into blocks of BLOCKX x BLOCKY threads */
#define BLOCKY 32
#define BLOCKX 32
#define MAXITER 255 // you may want to increase this

double xleft = -2.01;
double xright = 1;
double yupper, ylower;
double ycenter = 1e-6;
double step;

int host_pixel[XSIZE * YSIZE];
int device_pixel[XSIZE * YSIZE];

// NOTE: Had to switch from double to float due to precision errors on the GPU which uses FP32.
typedef struct {
    float real, imag;
} my_complex_t;

#define PIXEL(i, j) ((i) + (j) * XSIZE)

// ********** SUBTASK1: Create kernel device_calculate ******************/
__global__ void cuda_calculate(int *pixel, double xleft, double yupper, double step, int maxiter)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure we only calculate pixels within the image.
    // Right now every thread falls inside the image, but with varying image sizes I don't think
    // we can guarantee this.
    if (i > XSIZE || j > YSIZE) {
        // printf("Thread fell outside\n");
        return;
    }

    my_complex_t c, z, temp;
    int iter = 0;
    c.real = (xleft + step * i);
    c.imag = (yupper - step * j);
    z = c;
    while (z.real * z.real + z.imag * z.imag < 4.0) {
        temp.real = z.real * z.real - z.imag * z.imag + c.real;
        temp.imag = 2.0 * z.real * z.imag + c.imag;
        z = temp;
        if (++iter == maxiter)
            break;
    }
    //printf("%d\n", iter);
    pixel[PIXEL(i, j)] = iter;
}
// ********** SUBTASK1 END ***********************************************/

void host_calculate()
{
    for (int j = 0; j < YSIZE; j++) {
        for (int i = 0; i < XSIZE; i++) {
            /* Calculate the number of iterations until divergence for
            each pixel.
            If divergence never happens, return MAXITER */
            my_complex_t c, z, temp;
            int iter = 0;
            c.real = (xleft + step * i);
            c.imag = (yupper - step * j);
            z = c;
            while (z.real * z.real + z.imag * z.imag < 4.0) {
                temp.real = z.real * z.real - z.imag * z.imag + c.real;
                temp.imag = 2.0 * z.real * z.imag + c.imag;
                z = temp;
                if (++iter == MAXITER)
                    break;
            }
            host_pixel[PIXEL(i, j)] = iter;
        }
    }
}

typedef unsigned char uchar;
// save 24-bits bmp file, buffer must be in bmp format: upside-down
void savebmp(char *name, uchar *buffer, int x, int y)
{
    FILE *f = fopen(name, "wb");
    if (!f) {
        printf("Error writing image to disk.\n");
        return;
    }
    unsigned int size = x * y * 3 + 54;
	uchar header[54]={'B','M',size&255,(size>>8)&255,(size>>16)&255,size>>24,0,
		0,0,0,54,0,0,0,40,0,0,0,x&255,x>>8,0,0,y&255,y>>8,0,0,1,0,24,0,0,0,0,0,0,
		0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    fwrite(header, 1, 54, f);
    fwrite(buffer, 1, x * y * 3, f);
    fclose(f);
}

// given iteration number, set a color
void fancycolour(uchar *p, int iter)
{
    if (iter == MAXITER) {
        ;
    } else if (iter < 8) {
        p[0] = 128 + iter * 16;
        p[1] = p[2] = 0;
    } else if (iter < 24) {
        p[0] = 255;
        p[1] = p[2] = (iter - 8) * 16;
    } else if (iter < 160) {
        p[0] = p[1] = 255 - (iter - 24) * 2;
        p[2] = 255;
    } else {
        p[0] = p[1] = (iter - 160) * 2;
        p[2] = 255 - (iter - 160) * 2;
    }
}
// Get system time to microsecond precision
// ostensibly, similar to MPI_Wtime),
// returns time in seconds
double walltime(void)
{
    static struct timeval t;
    gettimeofday(&t, NULL);
    return (t.tv_sec + 1e-6 * t.tv_usec);
}


int main(int argc, char **argv)
{
    if (argc == 1) {
        puts("Usage: MANDEL n");
        puts("n decides whether image should be written to disk (1=yes,0=no)");
        return 0;
    }
    double start;
    double hosttime = 0;
    double devicetime = 0;
    double memtime = 0;

    hipDeviceProp_t p;
    hipSetDevice(0);
    hipGetDeviceProperties(&p, 0);
    printf("Device compute capability: %d.%d\n", p.major, p.minor);
    printf("Number of streaming multiprocessors: %d\n", p.multiProcessorCount);
    // The device major.minor is 7.5, meaning the GPU is from the Turing architecture
    // Turring GPU's have 64 cuda cores per streaming multiprocessor. The above line prints
    // that I get 40 SM's, meaning we get 64 * 40 = 2560 CUDA cores.

    /* Calculate the range in the y-axis such that we preserve the
    aspect ratio */
    step = (xright - xleft) / XSIZE;
    yupper = ycenter + (step * YSIZE) / 2;
    ylower = ycenter - (step * YSIZE) / 2;
    /* Host calculates image */
    start = walltime();
    host_calculate();
    hosttime += walltime() - start;

    //********** SUBTASK2: Set up device memory ***************************/
    int *device_alloced_pixel;
    hipMalloc(&device_alloced_pixel, XSIZE * YSIZE * sizeof(int));
    /********** SUBTASK2 END **********************************************/

    start = walltime();

    //********* SUBTASK3: Execute the kernel on the device ************/
    dim3 blockSize(BLOCKX, BLOCKY);
    dim3 gridSize((XSIZE + BLOCKX - 1) / BLOCKX, (YSIZE + BLOCKY - 1) / BLOCKY);
    cuda_calculate<<<gridSize, blockSize>>>(device_alloced_pixel, xleft, yupper, step, MAXITER);
    hipDeviceSynchronize(); // Called to ensure the kernel completes before measuring the time 
    //********** SUBTASK3 END *****************************************/

    devicetime += walltime() - start;
    start = walltime();

    //***** SUBTASK4: Transfer the result from device to device_pixel[][]*/
    hipError_t res = hipMemcpy(device_pixel, device_alloced_pixel, XSIZE * YSIZE * sizeof(int), hipMemcpyDeviceToHost);
    if (res != hipSuccess) {
        printf("CUDA memcpy error: %s\n", hipGetErrorString(res));
    }
    //********** SUBTASK4 END ******************************************/

    memtime += walltime() - start;

    /****** SUBTASK5: Free the device memory also ************************/
    hipFree(device_alloced_pixel);
    /********** SUBTASK5 END ******************************************/

    int errors = 0;
    /* check if result is correct */
    for (int i = 0; i < XSIZE; i++) {
        for (int j = 0; j < YSIZE; j++) {
            int diff = host_pixel[PIXEL(i, j)] - device_pixel[PIXEL(i, j)];
            if (diff < 0)
                diff = -diff;
            /* allow +-1 difference */
            if (diff > 1) {
                if (errors < 10) {
                    printf("Error on pixel %d %d: expected%d, found %d\n", i, j,
                           host_pixel[PIXEL(i, j)], device_pixel[PIXEL(i, j)]);
                } else if (errors == 10) {
                    puts("...");
                }
                errors++;
            }
        }
    }
    if (errors > 0) {
        printf("Found %d errors.\n", errors);
    } else {
        puts("Device calculations are correct.");
    }
    printf("\n");
    printf("Host time: %7.3f ms\n", hosttime * 1e3);
    printf("Device calculation: %7.3f ms\n", devicetime * 1e3);
    printf("Copy result: %7.3f ms\n", memtime * 1e3);


    if (strtol(argv[1], NULL, 10) != 0) {
        /* create nice image from iteration counts. take care to create it
        upside
        down (bmp format) */
        unsigned char *buffer = (unsigned char *)calloc(XSIZE * YSIZE * 3, 1);
        for (int i = 0; i < XSIZE; i++) {
            for (int j = 0; j < YSIZE; j++) {
                int p = ((YSIZE - j - 1) * XSIZE + i) * 3;
                fancycolour(buffer + p, device_pixel[PIXEL(i, j)]);
            }
        }
        /* write image to disk */
        savebmp("mandel1.bmp", buffer, XSIZE, YSIZE);
    }
    return 0;
}

#include "hip/hip_runtime.h"
#define _XOPEN_SOURCE 600
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <math.h>
#include <sys/time.h>


// TASK: T1
// Include the cooperative groups library
// BEGIN: T1
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;
// END: T1


// Convert 'struct timeval' into seconds in double prec. floating point
#define WALLTIME(t) ((double)(t).tv_sec + 1e-6 * (double)(t).tv_usec)

// Option to change numerical precision
typedef int64_t int_t;
typedef double real_t;

// TASK: T1b
// The three grids used for the calculations. Allocated on the device. 
real_t *d_prv;
real_t *d_cur;
real_t *d_nxt;
#define d_U_prv(i,j) d_prv[((i)+1)*(N+2)+(j)+1]
#define d_U(i,j)     d_cur[((i)+1)*(N+2)+(j)+1]
#define d_U_nxt(i,j) d_nxt[((i)+1)*(N+2)+(j)+1]

// The h_grid variable holds enough space for exactly one grid.
// Used for the initial condition, and to copy data from the device before storing to file.
real_t *h_grid;
#define h_U(i,j)     h_grid[((i)+1)*(N+2)+(j)+1]
// BEGIN: T1b
#define BLOCK_DIM_X 16
#define BLOCK_DIM_Y 16

// Simulation parameters: size, step count, and how often to save the state
int_t
    N = 128,
    M = 128,
    max_iteration = 1000000,
    snapshot_freq = 1000;

// Wave equation parameters, time step is derived from the space step
const real_t
    c  = 1.0,
    dx = 1.0,
    dy = 1.0;
real_t
    dt;

// END: T1b

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Rotate the time step buffers.
void move_buffer_window() {
    // Swap the pointers
    real_t* temp = d_prv;
    d_prv = d_cur;
    d_cur = d_nxt;
    d_nxt = temp;
}


// Save the present time step in a numbered file under 'data/'
void domain_save ( int_t step )
{
    char filename[256];
    sprintf ( filename, "data/%.5ld.dat", step );
    FILE *out = fopen ( filename, "wb" );
    if (out == NULL) {
        printf("ERROR: Could not open file %s\n", filename);
        exit(EXIT_FAILURE);
    }
    // TODO: Probably a way to write everything in one go.
    for ( int_t i=0; i<M; i++ )
    {
        fwrite ( &h_U(i,0), sizeof(real_t), N, out );
    }
    fclose ( out );
}


// TASK: T4
// Get rid of all the memory allocations
void domain_finalize ( void )
{
// BEGIN: T4
    free(h_grid);
    hipFree(d_prv);
    hipFree(d_cur);
    hipFree(d_nxt);
// END: T4
}


// TASK: T6
// Neumann (reflective) boundary condition
// BEGIN: T6
__device__ void boundary_condition_gpu(real_t *d_cur, int i, int j, int M, int N) {
    if (i == 0) {
        d_U(-1, j) = d_U(1, j);
    }
    if (i == M - 1) {
        d_U(M, j) = d_U(M-2, j);
    }

    if (j == 0) {
        d_U(i, -1) = d_U(i, 1);
    }
    if (j == N - 1) {
        d_U(i, N) = d_U(i, N-2);
    }
}

// TASK: T5
// Integration formula
// BEGIN; T5
__global__ void time_step_gpu(real_t *d_prv, real_t *d_cur, real_t *d_nxt, int M, int N, real_t dt, real_t c, real_t dx, real_t dy)
{
    // Indices of the thread in the grid
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Return if the thread is out of bounds
    if (i < 0 || i >= M || j < 0 || j >= N) {
        return;
    }

    boundary_condition_gpu(d_cur, i, j, M, N);

    d_U_nxt(i,j) = -d_U_prv(i,j) + 2.0*d_U(i,j)
                     + (dt*dt*c*c)/(dx*dy) * (
                        d_U(i-1,j)+d_U(i+1,j)+d_U(i,j-1)+d_U(i,j+1)-4.0*d_U(i,j)
                     );

    // Since each time step is dependent on calculations from the previous timetstep, we
    // need to ensure every computation of the current timestep is completed before any thread 
    // can continue onto the next timestep.
    cg::grid_group grid = cg::this_grid();
    grid.sync();
}

// TASK: T7
// Main time integration.
void simulate( void )
{
// BEGIN: T7
    dim3 blockDim(BLOCK_DIM_X, BLOCK_DIM_Y);
    dim3 gridDim(M / BLOCK_DIM_X, N / BLOCK_DIM_Y);

    void *args[] = {(void *)&d_prv, (void *)&d_cur, (void *)&d_nxt, (void *)&M, (void *)&N, (void *)&dt, (void *)&c, (void *)&dx, (void *)&dy};

    // Go through each time step
    for ( int_t iteration=0; iteration<=max_iteration; iteration++ )
    {
        if ( (iteration % snapshot_freq)==0 )
        {
            cudaErrorCheck(hipMemcpy(h_grid, d_cur, (M+2)*(N+2) * sizeof(real_t), hipMemcpyDeviceToHost));
            domain_save ( iteration / snapshot_freq );
        }

        // Perform the time step using on the GPU
        hipLaunchCooperativeKernel((void*)time_step_gpu, gridDim, blockDim, args);
        // time_step_gpu<<<gridDim, blockDim>>>(d_prv, d_cur, d_nxt, M, N, dt, c, dx, dy);
        // hipDeviceSynchronize();

        // Rotate the time step buffers
        move_buffer_window();
    }
// END: T7
}


// TASK: T8
// GPU occupancy
void occupancy( void )
{
// BEGIN: T8
    int threads_per_block = BLOCK_DIM_X * BLOCK_DIM_Y;
    int blocks_per_grid = (M / BLOCK_DIM_X) * (N / BLOCK_DIM_Y);

    hipDeviceProp_t p;
    hipGetDeviceProperties(&p, 0);

    int max_active_blocks_per_sm;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &max_active_blocks_per_sm,
        time_step_gpu,
        threads_per_block,
        0
    );

    int active_warps_per_sm = max_active_blocks_per_sm * (BLOCK_DIM_X * BLOCK_DIM_Y / 32);
    int max_warps_per_sm = p.maxThreadsPerMultiProcessor / 32;
    float occupancy = (float)active_warps_per_sm / (float)max_warps_per_sm;

    printf("--- Occupancy ---\n");
    printf("Grid size: %d\n", blocks_per_grid);
    printf("Launched blocks of size: %d\n", threads_per_block);
    printf("Theoretical occupancy: %f\n", occupancy);
// END: T8
}


// TASK: T2
// Make sure at least one CUDA-capable device exists
static bool init_cuda()
{
    hipDeviceProp_t p;
    int device_count = 0;
    if (hipGetDeviceCount(&device_count) != hipSuccess || device_count == 0) {
        printf("ERROR: No CUDA devices found!\n");
        return false;
    }

    hipSetDevice(0);
    hipGetDeviceProperties(&p, 0);

    if (!p.cooperativeLaunch) {
        printf("ERROR: Device does not support cooperative kernel launch!\n");
        return false;
    }

    printf("Name: %s\n", p.name);
    printf("Compute capability: %d.%d\n", p.major, p.minor);
    printf("Multiprocessors: %d\n", p.multiProcessorCount);
    printf("Warp size: %d\n", p.warpSize);
    return true;
}


// TASK: T3
// Set up our three buffers, and fill two with an initial perturbation
void domain_initialize ( void )
{
// BEGIN: T3
    bool locate_cuda = init_cuda();
    if (!locate_cuda)
    {
        exit( EXIT_FAILURE );
    }

    // Host data to store the state of a single grid
    h_grid = (real_t *) malloc ( (M+2)*(N+2)*sizeof(real_t) );

    // The three grids on the GPU we will use to do the calculations 
    cudaErrorCheck(hipMalloc(&d_prv, (M+2)*(N+2) * sizeof(real_t)));
    cudaErrorCheck(hipMalloc(&d_cur, (M+2)*(N+2) * sizeof(real_t)));
    cudaErrorCheck(hipMalloc(&d_nxt, (M+2)*(N+2) * sizeof(real_t)));

    for ( int_t i=0; i<M; i++ )
    {
        for ( int_t j=0; j<N; j++ )
        {
            // Calculate delta (radial distance) adjusted for M x N grid
            real_t delta = sqrt ( ((i - M/2.0) * (i - M/2.0)) / (real_t)M +
                                ((j - N/2.0) * (j - N/2.0)) / (real_t)N );
            h_U(i,j) = exp ( -4.0*delta*delta );
        }
    }

    // Send initial condition from host to device
    cudaErrorCheck(hipMemcpy(d_prv, h_grid, (M+2)*(N+2) * sizeof(real_t), hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(d_cur, h_grid, (M+2)*(N+2) * sizeof(real_t), hipMemcpyHostToDevice));

    // Set the time step for 2D case
    dt = dx*dy / (c * sqrt (dx*dx+dy*dy));
// END: T3
}


int main ( void )
{
    // Set up the initial state of the domain
    domain_initialize();

    struct timeval t_start, t_end;

    gettimeofday ( &t_start, NULL );
    simulate();
    gettimeofday ( &t_end, NULL );

    printf ( "Total elapsed time: %lf seconds\n",
        WALLTIME(t_end) - WALLTIME(t_start)
    );

    occupancy();

    // Clean up and shut down
    domain_finalize();
    exit ( EXIT_SUCCESS );
}
